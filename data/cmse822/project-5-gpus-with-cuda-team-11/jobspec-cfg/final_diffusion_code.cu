
#include <hip/hip_runtime.h>
#include <iostream>
#include <fstream>
#include <string>
#include <stdio.h>
#include <cmath>
#include <cassert>


#include <sys/time.h>

void get_walltime_(double* wcTime) {
  struct timeval tp;
  gettimeofday(&tp, NULL);
  *wcTime = (double)(tp.tv_sec + tp.tv_usec/1000000.0);
}

void get_walltime(double* wcTime) {
  get_walltime_(wcTime);
}

using namespace std;

const unsigned int NG = 2;

// Block sizes
//const unsigned int BLOCK_DIM_X = 128;
//const unsigned int BLOCK_DIM_X = 256;
//const unsigned int BLOCK_DIM_X = 512;
const unsigned int BLOCK_DIM_X = 1024;


__constant__ float c_a, c_b, c_c;

/******************************************************************************
  Error checking function for CUDA
 *****************************************************************************/
// Convenience function for checking CUDA runtime API results
// can be wrapped around any runtime API call. No-op in release builds.
//https://github.com/parallel-forall/code-samples/blob/master/series/cuda-cpp/
//        finite-difference/finite-difference.cu
inline
hipError_t checkCuda(hipError_t result)
{
#if defined(DEBUG) || defined(_DEBUG)
    if (result != cudaSuccess)
    {
        fprintf(stderr, "CUDA Runtime Error: %s\n",
                cudaGetErrorString(result));
        assert(result == cudaSuccess);
    }
#endif
    return result;
}

/******************************************************************************
  Do one diffusion step, on the host in host memory
 *****************************************************************************/
void host_diffusion(float* u, float *u_new, const unsigned int n,
                    const float dx, const float dt)
{
  //First, do the diffusion step on the interior points
    for(int i = NG; i < n-NG;i++){
        u_new[i] = u[i] + dt/(dx*dx) *(
                        - 1./12.f* u[i-2]
                        + 4./3.f * u[i-1]
                        - 5./2.f * u[i]
                        + 4./3.f * u[i+1]
                        - 1./12.f* u[i+2]);

   }

    //Apply the dirichlet boundary conditions
    u_new[0] = -u_new[NG+1];
    u_new[1] = -u_new[NG];

    u_new[n-NG]   = -u_new[n-NG-1];
    u_new[n-NG+1] = -u_new[n-NG-2];
}
/******************************************************************************
  Do one diffusion step, with CUDA
 *****************************************************************************/
__global__
void cuda_diffusion(float* u, float *u_new, const unsigned int n)
{
    //Do the diffusion
    //FIXME

    // get the global index in the 1-d grid of blocks which contain threads 
    int i = blockIdx.x*blockDim.x + threadIdx.x + 2;

    // constants already include the negative signs 
    u_new[i] = u[i] + ( c_a*u[i-2]
                       +c_b*u[i-1]
                       +c_c*u[i]
                       +c_b*u[i+1]
                       +c_a*u[i+2]);


    //Apply the dirichlet boundary conditions 

    if(i==2) { u_new[1] = -u_new[i] ; }
    if(i==3) { u_new[0] = -u_new[i];}

    if(i==n-3) {   u_new[n-2] = -u_new[i]; }
    if(i==n-4){    u_new[n-1] = -u_new[i];  }





}

/******************************************************************************
  Do one diffusion step, with CUDA, with shared memory
 *****************************************************************************/
__global__
void shared_diffusion(float* u, float *u_new, const unsigned int n)
{
    //Allocate the shared memory


    __shared__ float shared_m[BLOCK_DIM_X + 4];

    // local index for shared memory
    int s_index_ = threadIdx.x + 2;

    // global index of thread 
    int g_index_ = blockIdx.x*blockDim.x + s_index_;

    shared_m[s_index_] = u[g_index_];

    if(s_index_ < 4)
    {
         shared_m[s_index_ - 2] = u[g_index_ - 2];
    }
    else if(s_index_ >= blockDim.x - 4)
    {
       shared_m[s_index_ + 2] = u[g_index_ + 2];
    }
    __syncthreads();

    u_new[g_index_] = shared_m[s_index_] + ( c_a*shared_m[s_index_-2]
                          + c_b*shared_m[s_index_-1]
                          + c_c*shared_m[s_index_]
                          + c_b*shared_m[s_index_+1]
                          + c_a*shared_m[s_index_+2]);

    //Apply the dirichlet boundary conditions
    if(g_index_ < 4)
     {
        u_new[(g_index_ + 1)%2] = -u_new[g_index_];
     }
    if(g_index_ >= n - 4)
    {
     u_new[2*(n - 2)-(g_index_ + 1)] = -u_new[g_index_];
    }

}

/******************************************************************************
  Dump u to a file
 *****************************************************************************/
void outputToFile(string filename, float* u, unsigned int n)
{
    ofstream file;
    file.open(filename.c_str());
    file.precision(8);
    file << std::scientific;
    for(int i =0; i < n;i++){
        file<<u[i]<<endl;
    }
    file.close();
};

/******************************************************************************
  main
 *****************************************************************************/
int main(int argc, char** argv)
{
    //Number of steps to iterate
    //const unsigned int n_steps = 10;
    const unsigned int n_steps = 100;
    //const unsigned int n_steps = 1000000;

    //Whether and how ow often to dump data
    //const bool outputData = true;
    const bool outputData = false;
    const unsigned int outputPeriod = n_steps/10;

    //Size of u
    //const unsigned int n = (1<<11) +2*NG;
    //const unsigned int n = (1<<15) +2*NG;
    const unsigned int n = (1<<20) +2*NG;

    //Block and grid dimensions
    const unsigned int blockDim = BLOCK_DIM_X;
    const unsigned int gridDim = (n-2*NG)/blockDim;

    //Physical dimensions of the domain
    const float L = 2*M_PI;
    const float dx = L/(n-2*NG-1);
    const float dt = 0.25*dx*dx;

    //Create constants for 6th order centered 2nd derivative
    // NOTE: I added the appropriate negative signs here
    float const_a = -1./12.f * dt/(dx*dx);
    float const_b = 4./3.f  * dt/(dx*dx);
    float const_c = -5./2.f  * dt/(dx*dx);

    //Copy these the cuda constant memory
    //FIXME


    // Copy weights to device constants
    checkCuda(hipMemcpyToSymbol(HIP_SYMBOL(c_a), &const_a, sizeof(float),0, hipMemcpyHostToDevice));
    checkCuda(hipMemcpyToSymbol(HIP_SYMBOL(c_b), &const_b, sizeof(float),0, hipMemcpyHostToDevice));
    checkCuda(hipMemcpyToSymbol(HIP_SYMBOL(c_c), &const_c, sizeof(float),0, hipMemcpyHostToDevice));

    //iterator, for later
    int i;

    //Create cuda timers
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    //Timing variables
    float milliseconds;
    double startTime,endTime;

    //Filename for writing
    char filename[256];

    //Allocate memory for the initial conditions
    float* initial_u = new float[n];

    //Initialize with a periodic sin wave that starts after the left hand
    //boundaries and ends just before the right hand boundaries
    for( i = NG; i < n-NG; i++)
    {
        initial_u[i] = sin( 2*M_PI/L*(i-NG)*dx);
    }
    //Apply the dirichlet boundary conditions
    initial_u[0] = -initial_u[NG+1];
    initial_u[1] = -initial_u[NG];

    initial_u[n-NG]   = -initial_u[n-NG-1];
    initial_u[n-NG+1] = -initial_u[n-NG-2];

    /**************************************************************************
    Test the host kernel for diffusion
    **************************************************************************/

    //Allocate memory in the host's heap
    float* host_u  = new float[n];
    float* host_u2 = new float[n];//buffer used for u_new

    //Initialize the host memory
    for( i = 0; i < n; i++)
    {
        host_u[i] = initial_u[i];
    }

    outputToFile("data/host_uInit.dat",host_u,n);

    get_walltime(&startTime);
    //Perform n_steps of diffusion
    for( i = 0 ; i < n_steps; i++){

        if(outputData && i%outputPeriod == 0)
        {   printf("ok \n");
            sprintf(filename, "/mnt/home/hejaseba/mpii/cuda_/project_5/host_u%08d.dat", i);
            outputToFile(filename,host_u,n);
        }

        host_diffusion(host_u, host_u2, n, dx, dt);

        //Switch the buffer with the original u
        float* tmp = host_u;
        host_u = host_u2;
        host_u2 = tmp;

    }
    get_walltime(&endTime);

    cout << "Host function took: " << (endTime-startTime)*1000./n_steps
         << " ms per step" << endl;

    outputToFile("data/host_uFinal.dat", host_u, n);



  /**************************************************************************
    Test the cuda kernel for diffusion
    **************************************************************************/
    //Allocate a copy for the GPU memory in the host's heap
    float* cuda_u  = new float[n];

    //Initialize the cuda memory
    for( i = 0; i < n; i++)
    {
        cuda_u[i] = initial_u[i];
    }
    outputToFile("data/cuda_uInit.dat", cuda_u, n);

    //Allocate memory on the GPU
    float* d_u, *d_u2;
    //FIXME Allocate d_u,d_u2 on the GPU, and copy cuda_u into d_u

    // allocating data on the device's (gpu's) memory
    checkCuda(hipMalloc((void**)&d_u, n*sizeof(float)));
    checkCuda(hipMalloc((void**)&d_u2,n*sizeof(float)));

    // host to device copy
    checkCuda(hipMemcpy(d_u, cuda_u, n*sizeof(float),hipMemcpyHostToDevice));

    hipEventRecord(start);//Start timing
    //Perform n_steps of diffusion
    for( i = 0 ; i < n_steps; i++)
    {
        if(outputData && i%outputPeriod == 0)
        {
            //Copy data off the device for writing
            sprintf(filename, "/mnt/home/hejaseba/mpii/cuda_/project_5/cuda_u%08d.dat", i);
            //FIXME

            // device to host copy
            checkCuda(hipMemcpy(cuda_u, d_u, n*sizeof(float), hipMemcpyDeviceToHost));

            outputToFile(filename,cuda_u,n);
        }

        //Call the cuda_diffusion kernel
        //FIXME

        cuda_diffusion<<<gridDim, blockDim>>>(d_u, d_u2, n);

        //Switch the buffer with the original u
        //FIXME

        float* tmp = d_u;

        d_u = d_u2;
        d_u2 = tmp;
    }
    hipEventRecord(stop);//End timing

    //Copy the memory back for one last data dump
    sprintf(filename, "data/cuda_u%08d.dat", i);
    //FIXME


    // device to host copy solution
    checkCuda(hipMemcpy(cuda_u, d_u, n*sizeof(float),hipMemcpyDeviceToHost));
    outputToFile("data/cuda_uFinal.dat",cuda_u,n);

    //Get the total time used on the GPU
    hipEventSynchronize(stop);
    milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);

    cout << "Cuda Kernel took: " << milliseconds/n_steps
         << " ms per step" << endl;





   /**************************************************************************
    Test the cuda kernel for diffusion with shared memory
    **************************************************************************/

    //Allocate a copy for the GPU memory in the host's heap
    float* shared_u  = new float[n];

    //Initialize the cuda memory
    for( i = 0; i < n; i++)
    {
        shared_u[i] = initial_u[i];
    }
    outputToFile("data/shared_uInit.dat",shared_u,n);

    //Copy the initial memory onto the GPU
    //FIXME copy shared_u to d_u

    checkCuda(hipMemcpy(d_u, shared_u, n*sizeof(float),hipMemcpyHostToDevice));


    hipEventRecord(start);//Start timing
    //Perform n_steps of diffusion
    for( i = 0 ; i < n_steps; i++)
    {
        if(outputData && i%outputPeriod == 0)
        {
            //Copy data off the device for writing
            sprintf(filename,"/mnt/home/hejaseba/mpii/cuda_/project_5/shared_u%08d.dat",i);
            //FIXME




            checkCuda(hipMemcpy(shared_u, d_u, n*sizeof(float), hipMemcpyDeviceToHost));


            outputToFile(filename,shared_u,n);
        }


        shared_diffusion<<<gridDim, blockDim>>>(d_u, d_u2, n);



        float* tmp = d_u;
        d_u = d_u2;
        d_u2 = tmp;

    }
    hipEventRecord(stop);//End timing


    sprintf(filename,"data/shared_u%08d.dat",i);
    //FIXME

    // Copy final solution back to the device
    checkCuda(hipMemcpy(shared_u, d_u, n*sizeof(float),hipMemcpyDeviceToHost));

    //Get the total time used on the GPU
    hipEventSynchronize(stop);
    milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);

    cout << "Shared Memory Kernel took: " << milliseconds/n_steps
         << " ms per step" << endl;



    /**************************************************************************
    Test the cuda kernel for diffusion, with excessive memcpys
    **************************************************************************/

    //Initialize the cuda memory
    for( i = 0; i < n; i++)
    {
        shared_u[i] = initial_u[i];
    }

    hipEventRecord(start);//Start timing
    //Perform n_steps of diffusion
    for( i = 0 ; i < n_steps; i++)
    {

        checkCuda(hipMemcpy(d_u, shared_u, n*sizeof(float),hipMemcpyHostToDevice));

        shared_diffusion<<<gridDim, blockDim>>>(d_u, d_u2, n);

        checkCuda(hipMemcpy(shared_u, d_u2, n*sizeof(float),hipMemcpyDeviceToHost));
    }
    hipEventRecord(stop);//End timing

    //Get the total time used on the GPU
    hipEventSynchronize(stop);
    milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);

    cout << "Excessive hipMemcpy took: " << milliseconds/n_steps
         << " ms per step" << endl;

    // freeing memory on host
    delete[] initial_u;
    delete[] host_u;
    delete[] host_u2;

    delete[] cuda_u;
    delete[] shared_u;


    // freeing memory on device
    checkCuda( hipFree(d_u) );
    checkCuda( hipFree(d_u2) );
}
